#include "hspmd/core/ndarray.h"
#include "hspmd/impl/stream/CUDAStream.h"
#include "hspmd/impl/utils/common_utils.h"
#include "hspmd/impl/utils/cuda_utils.h"
#include "hspmd/impl/utils/cuda_math.h"
#include "hspmd/impl/utils/offset_calculator.cuh"
#include "hspmd/impl/kernel/Vectorized.cuh"

namespace hspmd {
namespace impl {

void PowCuda(const NDArray& input, double exponent, NDArray& output,
             const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_SAME_SHAPE(input, output);

  size_t size = output->numel();
  if (size == 0)
    return;
  HT_DISPATCH_FLOATING_TYPES(
    input->dtype(), spec_t, "PowCuda", [&]() {
      using InType = std::tuple<spec_t>;
      using OutType = thrust::tuple<spec_t>;
      launch_loop_kernel<InType, OutType>({input}, {output}, size, stream,
                                         [exponent] __device__ (spec_t x) -> spec_t {
                                           return hspmd::cuda::cuda_pow(x, static_cast<spec_t>(exponent));
                                         });
    });
  NDArray::MarkUsedBy({input, output}, stream);
}

} // namespace impl
} // namespace hspmd
