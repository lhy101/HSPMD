#include "hspmd/core/ndarray.h"
#include "hspmd/impl/cuda/CUDARand.h"
#include "hspmd/impl/stream/CUDAStream.h"
#include "hspmd/impl/random/CUDARandomState.h"
#include "hspmd/impl/utils/common_utils.h"
#include "hspmd/impl/utils/cuda_utils.h"
#include "hspmd/impl/utils/offset_calculator.cuh"
#include "hspmd/impl/kernel/Vectorized.cuh"
#include <mutex>

namespace hspmd {
namespace impl {

void DropoutCuda(const NDArray& input, double drop_rate, uint64_t seed,
                 NDArray& output, NDArray& mask, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_SAME_DEVICE(input, mask);
  HT_ASSERT_SAME_SHAPE(input, output);
  HT_ASSERT_SAME_SHAPE(input, mask);
  size_t size = input->numel();
  if (size == 0)
    return;
  CUDAStream hip_stream(stream);
  hspmd::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  auto rand_state = GetCUDARandomState(hip_stream.device_id(), seed, 4);
  HT_DISPATCH_FLOATING_TYPES(input->dtype(), spec_t, "DropoutCuda", [&]() {
    using InType = std::tuple<spec_t>;
    using OutType = thrust::tuple<spec_t, bool>;
    launch_loop_kernel_with_idx<InType, OutType>({input}, {output, mask}, size, stream,
      [drop_rate, rand_state] __device__ (int idx, spec_t input) -> thrust::tuple<spec_t, bool> {
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(rand_state.seed, idx, rand_state.offset, &state);
        float temp = hiprand_uniform(&state);
        bool keep_mask = (temp >= drop_rate);
        return thrust::tuple<spec_t, bool>(input * keep_mask / (1 - drop_rate), keep_mask);
      });
  });
}

void DropoutGradientCuda(const NDArray& grad, const NDArray& fw_mask,
                         double drop_rate, NDArray& output,
                         const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(grad);
  HT_ASSERT_SAME_DEVICE(grad, fw_mask);
  HT_ASSERT_SAME_DEVICE(grad, output);
  HT_ASSERT_SAME_SHAPE(grad, fw_mask);
  HT_ASSERT_SAME_SHAPE(grad, output);
  size_t size = output->numel();
  if (size == 0)
    return;
  HT_DISPATCH_FLOATING_TYPES(grad->dtype(), spec_t, "DropoutGradientCuda", [&]() {
    using InType = std::tuple<spec_t, bool>;
    using OutType = thrust::tuple<spec_t>;
    launch_loop_kernel<InType, OutType>({grad, fw_mask}, {output}, size, stream,
      [drop_rate] __device__ (spec_t grad, bool fw_mask) -> spec_t {
        return grad * fw_mask / (1 - drop_rate);
      });
  });
}

} // namespace impl
} // namespace hspmd
