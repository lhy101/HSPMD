#include "hspmd/core/ndarray.h"
#include "hspmd/impl/stream/CUDAStream.h"
#include "hspmd/impl/utils/common_utils.h"
#include "hspmd/impl/utils/cuda_utils.h"
#include "hspmd/impl/utils/offset_calculator.cuh"
#include "hspmd/impl/kernel/Vectorized.cuh"

namespace hspmd {
namespace impl {

void MaskedfillCuda(const NDArray& input, const NDArray& mask,
                    double val, NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, mask);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_SAME_SHAPE(input, output);

  size_t size = input->numel();
  if (size == 0)
    return;
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "MaskfillCuda", [&]() {
      using InType = std::tuple<spec_t, int64_t>;
      using OutType = thrust::tuple<spec_t>;
      launch_loop_kernel<InType, OutType>({input, mask}, {output}, size, stream,
                                         [val] __device__ (spec_t in, int64_t mask_) -> spec_t {
                                           return bool(mask_) ? static_cast<spec_t>(val) : in;
                                         });
  });
  NDArray::MarkUsedBy({input, mask, output}, stream);
}

} // namespace impl
} // namespace hspmd
